#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "types.cu"

__device__ int deviceStrlen(const char* str) {
    int len = 0;
    while(str[len] != '\0') {
        len++;
    }
    return len;
}

// Function to build an IPv4 packet with payload
__device__ IPv4Packet buildIPv4PacketWithPayload(uint32_t sourceAddress, uint32_t destinationAddress) {
    IPv4Packet packet;
    
    // Set some dummy values of the packet fields
    packet.version = 4;
    packet.headerLength = 5; // 5 * 32-bit words
    packet.typeOfService = 0;
    packet.totalLength = sizeof(IPv4Packet);
    packet.identification = 12345;
    packet.flagsAndFragmentOffset = 0;
    packet.timeToLive = 64;
    packet.protocol = 6; // TCP
    packet.headerChecksum = 0;
    packet.sourceAddress = sourceAddress;
    packet.destinationAddress = destinationAddress;
    
    // Set the payload data
    const char* payloadData = "Dummy payload data";
    memcpy(packet.payload, payloadData, deviceStrlen(payloadData));
    
    return packet;
}

// Kernel function to generate IPv4 packets
__device__ uint32_t randomizeAddress(hiprandState_t* state) {
    uint32_t address = 0;
    for (int i = 0; i < 4; i++) {
        address |= (hiprand(state) % 256) << (i * 8); 
    }
    return address;
}

__global__ void generatePackets(IPv4Packet* packets, int numPackets) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numPackets) {
        hiprandState_t state;
        hiprand_init(clock64(), idx, 0, &state); // Initialize random number generator
        
        uint32_t sourceAddress = randomizeAddress(&state); // Generate random source address
        uint32_t destinationAddress = randomizeAddress(&state); // Generate random destination address
        
        packets[idx] = buildIPv4PacketWithPayload(sourceAddress, destinationAddress);
    }
}
