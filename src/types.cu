// file for data types

#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>

#ifndef IPV4_PACKET_H
#define IPV4_PACKET_H

struct IPv4Packet {
    uint8_t version;
    uint8_t headerLength;
    uint8_t typeOfService;
    uint16_t totalLength;
    uint16_t identification;
    uint16_t flagsAndFragmentOffset;
    uint8_t timeToLive;
    uint8_t protocol;
    uint16_t headerChecksum;
    uint32_t sourceAddress;
    uint32_t destinationAddress;
    uint8_t payload[1500]; // Maximum payload size of 1500 bytes
};

#endif // IPV4_PACKET_H